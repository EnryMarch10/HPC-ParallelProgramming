/****************************************************************************
 *
 * cuda-stencil1d.cu - 1D stencil example with CUDA
 *
 * Based on the examples from the CUDA toolkit documentation
 * http://docs.nvidia.com/cuda/cuda-c-programming-guide/
 *
 * Last updated in 2017 by Moreno Marzolla <https://www.moreno.marzolla.name/>
 *
 * ---------------------------------------------------------------------------
 *
 * This implementation does not use shared memory.
 *
 * Compile with:
 *
 *      nvcc cuda-stencil1d.cu -o cuda-stencil1d
 *
 * Run with:
 *
 *      ./cuda-stencil1d
 *
 ****************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLKDIM 1024
#define RADIUS 3

/* Size of the input EXCLUDING the first and last RADIUS elements */
#define N (BLKDIM * 1024)

__global__ void stencil1d(int *in, int *out) 
{
    const int index = threadIdx.x + blockIdx.x * blockDim.x + RADIUS;
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++) {
        result += in[index + offset];
    }
    /* Store the result */
    out[index] = result;
}

int main(void) 
{
    int *h_in, *h_out;	  /* host copies of in and out */
    int *d_in, *d_out;	  /* device copies of in and out */
    int i;
    const size_t size = (N + 2 * RADIUS) * sizeof(int); /* input size */

    assert(N % BLKDIM == 0);

    /* Allocate space for device copies of d_in and d_out */
    hipMalloc((void **) &d_in, size);
    hipMalloc((void **) &d_out, size);
    /* Allocate space for host copies of h_in and h_out */
    h_in = (int *) malloc(size);
    h_out = (int *) malloc(size);
    /* Set all elements of h_in to one */
    for (i = 0; i < N + 2 * RADIUS; i++) {
        h_in[i] = 1;
    }
    /* Copy input to device */
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
    /* Launch stencil1d() kernel on GPU */
    stencil1d<<<(N + BLKDIM - 1) / BLKDIM, BLKDIM>>>(d_in, d_out);
    /* Copy result back to host */
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
    /* Check the result */
    for (i = RADIUS; i < N + RADIUS; i++) {
        if (h_out[i] != 7) {
            fprintf(stderr, "Error at index %d: h_out[%d] == %d, expected 7\n", i, i, h_out[i]);
            return EXIT_FAILURE;
        }
    }
    printf("Test OK\n");
    /* Cleanup */
    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);
    return EXIT_SUCCESS;
}
