/****************************************************************************
 *
 * cuda-vecadd2.cu - Sum two arrays with CUDA, using thread
 *
 * Based on the examples from the CUDA toolkit documentation
 * http://docs.nvidia.com/cuda/cuda-c-programming-guide/
 *
 * Last updated in 2017 by Moreno Marzolla <https://www.moreno.marzolla.name/>
 *
 * ---------------------------------------------------------------------------
 *
 * Compile with:
 *
 *      nvcc cuda-vecadd2.cu -o cuda-vecadd2
 *
 * Run with:
 *
 *      ./cuda-vecadd2
 *
 ****************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c)
{
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void vec_init(int *a, int n)
{
    for (int i = 0; i < n; i++) {
        a[i] = i;
    }
}

#define N 1024

int main(void)
{
    int *a, *b, *c;	          /* host copies of a, b, c */
    int *d_a, *d_b, *d_c;	  /* device copies of a, b, c */
    int i;

    const size_t size = N * sizeof(int);
    /* Allocate space for device copies of a, b, c */
    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);
    /* Allocate space for host copies of a, b, c */
    a = (int *) malloc(size);
    vec_init(a, N);
    b = (int *) malloc(size);
    vec_init(b, N);
    c = (int *) malloc(size);
    /* Copy inputs to device */
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    /* Launch add() kernel on GPU */
    printf("Adding %d elements\n", N);
    add<<<1, N>>>(d_a, d_b, d_c);
    /* Copy result back to host */
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    /* Check results */
    for (i = 0; i < N; i++) {
        if (c[i] != a[i] + b[i]) {
            fprintf(stderr, "Error at index %d: a[%d]=%d, b[%d]=%d, c[%d]=%d\n",
                    i, i, a[i], i, b[i], i, c[i]);
            break;
        }
    }
    if (i == N) {
        printf("Check OK\n");
    }
    /* Cleanup */
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return EXIT_SUCCESS;
}
