#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "hpc.h"

#define BLKDIM 1024

typedef unsigned char cell_t;

/**
 * Given the current state of the CA, compute the next state.  This
 * version requires that the `cur` and `next` arrays are extended with
 * ghost cells; therefore, `ext_n` is the length of `cur` and `next`
 * _including_ ghost cells.
 *
 *                             +----- ext_n-2
 *                             |   +- ext_n-1
 *   0   1                     V   V
 * +---+-------------------------+---+
 * |///|                         |///|
 * +---+-------------------------+---+
 *
 */
__global__ void rule30_step(cell_t *cur, cell_t *next, int ext_n) {
    const int i = 1 + threadIdx.x + blockIdx.x * blockDim.x;

    if (i < ext_n - 1) {
        const cell_t left   = cur[i - 1];
        const cell_t center = cur[i];
        const cell_t right  = cur[i + 1];
        next[i] =
            ( left && !center && !right) ||
            (!left && !center &&  right) ||
            (!left &&  center && !right) ||
            (!left &&  center &&  right);
    }
}

/**
 * Initialize the domain; all cells are 0, with the exception of a
 * single cell in the middle of the domain. `cur` points to an array
 * of length `ext_n`; the length includes two ghost cells.
 */
void init_domain(cell_t *cur, int ext_n)
{
    for (int i = 0; i < ext_n; i++) {
        cur[i] = 0;
    }
    cur[ext_n / 2] = 1;
}

/**
 * Dump the current state of the CA to PBM file `out`. `cur` points to
 * an array of length `ext_n` that includes two ghost cells.
 */
void dump_state(FILE *out, const cell_t *cur, int ext_n)
{
    const int LEFT = 1;
    const int RIGHT = ext_n - 2;
    for (int i = LEFT; i <= RIGHT; i++) {
        fprintf(out, "%d ", cur[i]);
    }
    fprintf(out, "\n");
}

int main(int argc, char *argv[])
{
    const char *outname = "cuda-rule30.pbm";
    FILE *out;
    int width = 1024, steps = 1024;
    cell_t *cur;

    if (argc > 3) {
        fprintf(stderr, "Usage: %s [width [steps]]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if (argc > 1) {
        width = atoi(argv[1]);
    }

    if (argc > 2) {
        steps = atoi(argv[2]);
    }

    const int ext_width = width + 2;
    const size_t ext_size = ext_width * sizeof(*cur); /* includes ghost cells */
    const int LEFT_GHOST = 0;
    const int LEFT = 1;
    const int RIGHT_GHOST = ext_width - 1;
    const int RIGHT = RIGHT_GHOST - 1;
    /* Create the output file */
    out = fopen(outname, "w");
    if (!out) {
        fprintf(stderr, "FATAL: cannot create file \"%s\"\n", outname);
        return EXIT_FAILURE;
    }
    fprintf(out, "P1\n");
    fprintf(out, "# produced by cuda-rule30.cu\n");
    fprintf(out, "%d %d\n", width, steps);

    /* Allocate space for the `cur[]` and `next[]` arrays */
    cur = (cell_t *) malloc(ext_size);
    assert(cur != NULL);

    /* Initialize the domain */
    init_domain(cur, ext_width);

    cell_t *d_cur, *d_next;

    cudaSafeCall(hipMalloc((void **) &d_cur, ext_width));
    cudaSafeCall(hipMalloc((void **) &d_next, ext_width));

    /* Evolve the CA */
    for (int s = 0; s < steps; s++) {
        /* Dump the current state */
        dump_state(out, cur, ext_width);

        /* Fill ghost cells */
        cur[RIGHT_GHOST] = cur[LEFT];
        cur[LEFT_GHOST] = cur[RIGHT];

        cudaSafeCall(hipMemcpy(d_cur, cur, ext_width, hipMemcpyHostToDevice));
        /* Compute next state */
        rule30_step<<<(width + BLKDIM - 1) / BLKDIM, BLKDIM>>>(d_cur, d_next, ext_width);
        cudaCheckError();

        cudaSafeCall(hipMemcpy(cur, d_next, ext_width, hipMemcpyDeviceToHost));
    }

    cudaSafeCall(hipFree(d_cur));
    cudaSafeCall(hipFree(d_next));

    free(cur);

    fclose(out);

    return EXIT_SUCCESS;
}
